#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <assert.h>
#include "png_util.h"
#define MAX_N 20000

char plate[2][(MAX_N + 2) * (MAX_N + 2)];
char* d_plate[2][(MAX_N + 2) * (MAX_N + 2)];
int* d_which;
int which = 0;
int n;
int live(int index, int* plate){
    return (plate[index - n - 3] 
        + plate[index - n - 2]
        + plate[index - n - 1]
        + plate[index - 1]
        + plate[index + 1]
        + plate[index + n + 1]
        + plate[index + n + 2]
        + plate[index + n + 3]);
}
__global__ void iteration(int * d_which, int** d_plate) {
    int index = threadIdx.x * (blockDim.x + 2) + threadIdx.y;
    int num = live(index, d_plate[*d_which]);
    if(d_plate[*d_which][index]){
        d_plate[!*d_which][index] = (num == 2 || num == 3) ? 1 : 0;
    }else{
        d_plate[!*d_which][index] = (num == 3);
    }
}
void print_plate(){
    if (n < 60) {
        for(int i = 1; i <= n; i++){
            for(int j = 1; j <= n; j++){
                printf("%d", (int) plate[which][i * (n + 2) + j]);
            }
            printf("\n");
        }
    } else {
        printf("Plate too large to print to screen\n");
    }
    printf("\0");
}

void plate2png(char* filename) {
    char * img = (char *) malloc(n*n*sizeof(char));

    image_size_t sz;
    sz.width = n;
    sz.height = n; 

    for(int i = 1; i <= n; i++){
        for(int j = 1; j <= n; j++){
            int pindex = i * (n + 2) + j;
            int index = (i-1) * (n) + j;
            if (plate[!which][pindex] > 0)
                img[index] = 255; 
            else 
                img[index] = 0;
        }
    }
    printf("Writing file\n");
    write_png_file(filename,(unsigned char *) img,sz);

    printf("done writing png\n"); 
    free(img);
    printf("done freeing memory\n");
}

int main() { 
    int M;
    char line[MAX_N];
    if(scanf("%d %d", &n, &M) == 2){
        if (n > 0) {
            memset(plate[0], 0, sizeof(char) * (n + 2) * (n + 2));
            memset(plate[1], 0, sizeof(char) * (n + 2) * (n + 2));
            for(int i = 1; i <= n; i++){
                scanf("%s", &line);
                for(int j = 0; j < n; j++){
                    plate[0][i * (n + 2) + j + 1] = line[j] - '0';
                }
            }
        } else {
            n = MAX_N; 
            for(int i = 1; i <= n; i++) 
                for(int j = 0; j < n; j++) 
                    plate[0][i * (n+2) +j + 1] = (char) rand() % 2;
        }

        hipMalloc((void**)&d_plate, sizeof(char)*(n + 2)*(n + 2)*2);
        hipMalloc((void**)&d_which, sizeof(int));
        dim3 grid_size(1); dim3 block_size(n + 2, n + 2);
        for(int i = 0; i < M; i++){
            printf("\nIteration %d:\n",i);
            print_plate();
            hipMemcpy(d_plate, plate, sizeof(char)*(n + 2)*(n + 2)*2, hipMemcpyHostToDevice);
            hipMemcpy(d_which, &which, sizeof(int), hipMemcpyHostToDevice);
            iteration<<<grid_size, block_size>>>(d_which, &d_plate);
            hipMemcpy(plate, d_plate, sizeof(char)*(n + 2)*(n + 2)*2, hipMemcpyDeviceToHost);
            which=!which;
        }
        printf("\n\nFinal:\n");
        plate2png("plate.png");
        print_plate();
    }
    return 0;
}
