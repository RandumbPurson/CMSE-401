#include "hip/hip_runtime.h"
#include "png_util.h"
#include <assert.h>
#include <iterator>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <iostream>
#define CUDA_CALL(x)                                                           \
  {                                                                            \
    hipError_t cuda_error__ = (x);                                            \
    if (cuda_error__)                                                          \
      std::cout << "CUDA error: " << #x << " returned "                        \
                << hipGetErrorString(cuda_error__) << std::endl;              \
  }

#define MAX_N 20000
#define BLOCK_SIZE 20

char plate[2][(MAX_N + 2) * (MAX_N + 2)];
char *d_plate;
int which = 0;
int *d_which;
int n;
int *d_n;

__device__ int live(char *d_plate, int index, int n) {
  return (d_plate[index - n - 3] + d_plate[index - n - 2] +
          d_plate[index - n - 1] + d_plate[index - 1] + d_plate[index + 1] +
          d_plate[index + n + 1] + d_plate[index + n + 2] +
          d_plate[index + n + 3]);
}
__global__ void iteration(char *d_plate, int *d_which, int *d_n) {
  int x = (BLOCK_SIZE * blockIdx.x) + threadIdx.x + 1;
  int y = (BLOCK_SIZE * blockIdx.y) + threadIdx.y + 1;
  if (x >= (*d_n + 2) || y >= (*d_n + 2)) {
    return;
  }
  int index = (*d_n + 2) * y + x;

  int srcIdx = (*d_which) * (MAX_N + 2) * (MAX_N + 2) + index;
  int dstIdx = (!*d_which) * (MAX_N + 2) * (MAX_N + 2) + index;

  int num = live(d_plate, srcIdx, *d_n);
  if (d_plate[srcIdx]) {
    d_plate[dstIdx] = (num == 2 || num == 3) ? 1 : 0;
  } else {
    d_plate[dstIdx] = (num == 3);
  }
}
void print_plate() {
  if (n < 60) {
    for (int i = 1; i <= n; i++) {
      for (int j = 1; j <= n; j++) {
        printf("%d", (int)plate[which][i * (n + 2) + j]);
      }
      printf("\n");
    }
  } else {
    printf("Plate too large to print to screen\n");
  }
  printf("\0");
}

void plate2png(char *filename) {
  char *img = (char *)malloc(n * n * sizeof(char));

  image_size_t sz;
  sz.width = n;
  sz.height = n;

  for (int i = 1; i <= n; i++) {
    for (int j = 1; j <= n; j++) {
      int pindex = i * (n + 2) + j;
      int index = (i - 1) * (n) + j;
      if (plate[!which][pindex] > 0)
        img[index] = 255;
      else
        img[index] = 0;
    }
  }
  printf("Writing file\n");
  write_png_file(filename, (unsigned char *)img, sz);

  printf("done writing png\n");
  free(img);
  printf("done freeing memory\n");
}

int main() {
  int M;
  char line[MAX_N];
  if (scanf("%d %d", &n, &M) == 2) {
    if (n > 0) {
      memset(plate[0], 0, sizeof(char) * (n + 2) * (n + 2));
      memset(plate[1], 0, sizeof(char) * (n + 2) * (n + 2));
      for (int i = 1; i <= n; i++) {
        scanf("%s", &line);
        for (int j = 0; j < n; j++) {
          plate[0][i * (n + 2) + j + 1] = line[j] - '0';
        }
      }
    } else {
      n = MAX_N;
      for (int i = 1; i <= n; i++)
        for (int j = 0; j < n; j++)
          plate[0][i * (n + 2) + j + 1] = (char)rand() % 2;
    }

    CUDA_CALL(hipMalloc((void **)&d_plate,
                         sizeof(char) * 2 * (MAX_N + 2) * (MAX_N + 2)));
    CUDA_CALL(hipMalloc((void **)&d_which, sizeof(int)));
    CUDA_CALL(hipMalloc((void **)&d_n, sizeof(int)));
    CUDA_CALL(hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice));

    dim3 grid_size(MAX_N / BLOCK_SIZE, MAX_N / BLOCK_SIZE);
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    for (int i = 0; i < M; i++) {
      printf("\nIteration %d:\n", i);
      print_plate();
      CUDA_CALL(hipMemcpy(d_plate, &plate,
                           sizeof(char) * 2 * (MAX_N + 2) * (MAX_N + 2),
                           hipMemcpyHostToDevice));
      CUDA_CALL(
          hipMemcpy(d_which, &which, sizeof(int), hipMemcpyHostToDevice));
      iteration<<<grid_size, block_size>>>(d_plate, d_which, d_n);
      CUDA_CALL(hipMemcpy(&plate, d_plate,
                           sizeof(char) * 2 * (MAX_N + 2) * (MAX_N + 2),
                           hipMemcpyDeviceToHost));
      which = !which;
    }
    printf("\n\nFinal:\n");
    plate2png("plate.png");
    print_plate();
  }
  return 0;
}
